#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include<fstream>
#include<cstdlib>
#include<chrono>
#include<thrust/device_vector.h>
using namespace std;


__global__ void brute(const char *v,const char *s,int *d,int *max,int *count){
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    count[blockIdx.x]=0;
    __shared__ int  countl[1024]; 
    int j=0;
    while((j<(*d))&&(i<(*max))&&(v[i]==s[j]))
    {
    i++;    
   j++;
    }
    
    if(j==(*d))
    countl[threadIdx.x]=1;
    else
    countl[threadIdx.x]=0;
    __syncthreads();    
    for(i=0;i<blockDim.x;i++)
    count[blockIdx.x]=count[blockIdx.x]+countl[i];


}


int main(int argc,char **argv){
    int max,*d_max;    
    max=5000000;
    vector<string> v(1000);
    int *d_f,i,l1,j=0,num_blocks=max/1024+1,tot=0;
    char *d_str,*str;
    str=new char[max];
    char *d_cmp;
    int *d_count,*count;
    count=new int[num_blocks];
    for(i=0;i<num_blocks;i++)
    count[i]=0;
    char cmp[]="the";
    int f=sizeof(cmp)/sizeof(cmp[0])-1;
    ifstream fil("inp.txt"); 
    string line;
    while(getline(fil,line)&&j<max)
    {   //cout<<line;
        v.push_back(line);
        l1=line.length();
        for(i=0;i<l1&&j<max;i++)
        str[j++]=line[i];

    }
  //  cout<<"herre";
 //   cout<<f;
  //  for(i=0;i<max;i++)
   // cout<<str[i];
  // brute(str);
    hipMalloc((void**)&d_str,sizeof(char)*max);
    hipMalloc((void**)&d_cmp,sizeof(char)*f);
    hipMalloc((void**)&d_f,sizeof(int));
    hipMalloc((void**)&d_max,sizeof(int));
    hipMalloc((void**)&d_count,sizeof(int)*num_blocks);

    hipMemcpy(d_count,count,sizeof(int)*num_blocks,hipMemcpyHostToDevice);  
    hipMemcpy(d_f,&f,sizeof(int),hipMemcpyHostToDevice);  
    hipMemcpy(d_cmp,&cmp,sizeof(char)*f,hipMemcpyHostToDevice);  
    hipMemcpy(d_max,&max,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_str,str,max*sizeof(char),hipMemcpyHostToDevice);
 // auto start_time = std::chrono::high_resolution_clock::now();

    brute<<<num_blocks,1024>>>(d_str,d_cmp,d_f,d_max,d_count);
    
    hipDeviceSynchronize();
    hipFree(d_str);
    hipFree(d_cmp);
    hipFree(d_max);
    hipMemcpy(count,d_count,sizeof(int)*num_blocks,hipMemcpyDeviceToHost);
        for(i=0;i<num_blocks;i++){
        tot+=count[i];
        }

    hipFree(d_count);
    cout<<tot<<endl;
    delete[] count;
    delete[] str;
 // auto end_time = std::chrono::high_resolution_clock::now();
   // auto time = end_time - start_time;

  // cout<<std::chrono::duration_cast<std::chrono::milliseconds>(time).count()<<" to run.\n";

    return 0;

}